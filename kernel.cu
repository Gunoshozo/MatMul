#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include ""
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
using namespace std;


#define BLOCK_SIZE 32


__global__ void kernel(int* a, int* b, int* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int dot_prod = 0;

	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			dot_prod += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = dot_prod;
	}
}

void MatMulCPU(int* a, int* b, int* c, int m, int n, int k) {
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < k; j++)
		{
			int tmp_dot_prod = 0;
			for (int h = 0; h < n; h++)
			{
				tmp_dot_prod += a[i * n + h] * b[h * k + j];
			}
			c[i * k + j] = tmp_dot_prod;
		}
	}
}

void RandomFillMatrix(int* a, int m, int n) {
	for (int i = 0; i < m; i++)
		for (int j = 0; j < n; j++) {
			a[i * m + j] = rand();
		}
}

bool VerifyMatricies(int* a, int* b, int m, int n) {
	for (int i = 0; i < m; i++)
		for (int j = 0; j < n; j++) {
			if (a[i * m + j] != b[i * m + j])
				return false;
		}
	return true;
}

int main(int argc, char const* argv[])
{
	
	//matricies dimentions
	int m, n, k;
	//host allocated variables
	int* h_a, * h_b, * h_c, * h_dev_c_copy;
	//device allocaterd variables
	int* d_a, * d_b, * d_c;
	//variables for time measuremts
	float cpuTime, gpuTime;
	hipEvent_t start, stop;

	printf("Please type in matrices dimentions\n");
	printf("m: ");
	scanf("%d", &m);
	printf("n: ");
	scanf("%d", &n);
	printf("k: ");
	scanf("%d", &k);

	//memory allocation on the host
	hipHostMalloc((void**)&h_a, sizeof(int) * m * n);
	hipHostMalloc((void**)&h_b, sizeof(int) * n * k);
	hipHostMalloc((void**)&h_c, sizeof(int) * m * k);
	hipHostMalloc((void**)&h_dev_c_copy, sizeof(int) * m * k);

	//fill matricies with random numbers
	srand(time(NULL));
	RandomFillMatrix(h_a, m, n);
	RandomFillMatrix(h_b, n, k);

	//events for time time measurment on gpu
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//memory allocation on the device
	hipMalloc((void**)&d_a, sizeof(int) * m * n);
	hipMalloc((void**)&d_b, sizeof(int) * n * k);
	hipMalloc((void**)&d_c, sizeof(int) * m * k);
	
	//copy from host to device
	hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

	//init dimentions for kernel
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	//run the kernel
	kernel <<<dimGrid, dimBlock>>> (d_a, d_b, d_c, m, n, k);

	//copy results from device to host to verify later
	hipMemcpy(h_dev_c_copy, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//time measurment
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	auto begin = chrono::high_resolution_clock::now();

	MatMulCPU(h_a, h_b, h_c, m, n, k);

	auto end = chrono::high_resolution_clock::now();

	bool verify = VerifyMatricies(h_dev_c_copy, h_c, m, k);

	chrono::microseconds durationMs = chrono::duration_cast<chrono::milliseconds>(end - begin);
	cpuTime = durationMs.count();

	printf("\n============================\n");
	printf("Results:\n");
	printf("MultVerify: %s\n", verify ? "true" : "false");
	printf("CPU time: %f ms.\n", cpuTime);
	printf("GPU time: %f ms.\n", gpuTime);

	//freeing the allocated memory
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	hipHostFree(h_dev_c_copy);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}